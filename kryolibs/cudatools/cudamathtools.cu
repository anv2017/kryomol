#include "hip/hip_runtime.h"
/*****************************************************************************************
                            cudamathtools.cpp  -  description
                             -------------------
This file is part of the KryoMol project.
For more information, see <http://kryomol.sourceforge.io/>

This program is free software; you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation version 2 of the License.
******************************************************************************************/
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cudamathtools.h"

#ifndef M_PI
#define float M_PI 3.14159265358979323846;
#endif

#ifndef M_N
#define int M_N 256;
#endif


//Kernel functions
__global__ void VecAdd(float* a, float* b, float* c, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        c[i] = a[i]+b[i];
}

__global__ void VecSubs(float *a, float *b, float *c, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        c[i] = a[i]-b[i];

}

__global__ void ScMult(float w, float *a, float *b, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        b[i] = w*a[i];

}

__global__ void VecMult(float *a, float *b, float *c, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        c[i] = a[i]*b[i];

}

__global__ void Hamard(float *a, float *b, float *c, float *d, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] = a[i]*b[i]+c[i];
}

__global__ void SquareDiff(float w, float  *a, float *b, float *c, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        c[i] += w*(a[i]*a[i]-b[i]*b[i]);
}

__global__ void OrbitalS(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)));
}

__global__ void OrbitalPx(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx);
}

__global__ void OrbitalPy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(b[i]-cy);
}

__global__ void OrbitalPz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(c[i]-cz);
}

__global__ void OrbitalDxx(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx)*(a[i]-cx);
}

__global__ void OrbitalDxy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx)*(b[i]-cy);
}

__global__ void OrbitalDxz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx)*(c[i]-cz);
}

__global__ void OrbitalDyy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(b[i]-cy)*(b[i]-cy);
}

__global__ void OrbitalDyz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(b[i]-cy)*(c[i]-cz);
}

__global__ void OrbitalDzz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(c[i]-cz)*(c[i]-cz);
}

__global__ void OrbitalFxxx(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx)*(a[i]-cx)*(a[i]-cx);
}

__global__ void OrbitalFxxy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx)*(a[i]-cx)*(b[i]-cy);
}

__global__ void OrbitalFxxz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx)*(a[i]-cx)*(c[i]-cz);
}

__global__ void OrbitalFxyy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx)*(b[i]-cy)*(b[i]-cy);
}

__global__ void OrbitalFxyz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx)*(b[i]-cy)*(c[i]-cz);
}

__global__ void OrbitalFxzz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(a[i]-cx)*(c[i]-cz)*(c[i]-cz);
}

__global__ void OrbitalFyyy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(b[i]-cy)*(b[i]-cy)*(b[i]-cy);
}

__global__ void OrbitalFyyz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(b[i]-cy)*(b[i]-cy)*(c[i]-cz);
}

__global__ void OrbitalFyzz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(b[i]-cy)*(c[i]-cz)*(c[i]-cz);
}

__global__ void OrbitalFzzz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(c[i]-cz)*(c[i]-cz)*(c[i]-cz);
}

__global__ void OrbitalDY0(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(0.25)*(sqrt(15/(M_PI)))*(2*(c[i]-cz)*(c[i]-cz)-(a[i]-cx)*(a[i]-cx)-(b[i]-cy)*(b[i]-cy));
}

__global__ void OrbitalDY1(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(-0.5)*(sqrt(15/(2*M_PI)))*(c[i]-cz)*(a[i]-cx);
}

__global__ void OrbitalDY2(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(0.5)*(sqrt(15/(2*M_PI)))*(c[i]-cz)*(a[i]-cx);
}

__global__ void OrbitalDY3(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(0.25)*(sqrt(15/(2*M_PI)))*((a[i]-cx)*(a[i]-cx)-(b[i]-cy)*(b[i]-cy));
}

__global__ void OrbitalDY4(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(0.25)*(sqrt(15/(2*M_PI)))*((a[i]-cx)*(a[i]-cx)-(b[i]-cy)*(b[i]-cy));
}

__global__ void OrbitalFY0(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(0.25)*(sqrt(7/(M_PI)))*(c[i]-cz)*(2*(c[i]-cz)*(c[i]-cz)-3*(a[i]-cx)*(a[i]-cx)-3*(b[i]-cy)*(b[i]-cy));
}

__global__ void OrbitalFY1(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(-0.125)*(sqrt(21/(M_PI)))*(a[i]-cx)*(4*(c[i]-cz)*(c[i]-cz)-(a[i]-cx)*(a[i]-cx)-(b[i]-cy)*(b[i]-cy));
}

__global__ void OrbitalFY2(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(0.125)*(sqrt(21/(M_PI)))*(a[i]-cx)*(4*(c[i]-cz)*(c[i]-cz)-(a[i]-cx)*(a[i]-cx)-(b[i]-cy)*(b[i]-cy));
}

__global__ void OrbitalFY3(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(0.25)*(sqrt(105/(2*M_PI)))*(c[i]-cz)*((a[i]-cx)*(a[i]-cx)-(b[i]-cy)*(b[i]-cy));
}

__global__ void OrbitalFY4(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(0.25)*(sqrt(105/(2*M_PI)))*(c[i]-cz)*((a[i]-cx)*(a[i]-cx)-(b[i]-cy)*(b[i]-cy));
}

__global__ void OrbitalFY5(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(-0.125)*(sqrt(35/M_PI))*(a[i]-cx)*((a[i]-cx)*(a[i]-cx)-(b[i]-cy)*(b[i]-cy));
}

__global__ void OrbitalFY6(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i<N)
        d[i] += xs*exp(-((a[i]-cx)*(a[i]-cx)+(b[i]-cy)*(b[i]-cy)+(c[i]-cz)*(c[i]-cz)))*(0.125)*(sqrt(35/M_PI))*(a[i]-cx)*((a[i]-cx)*(a[i]-cx)-(b[i]-cy)*(b[i]-cy));
}


//Cuda functions
void scudaadd(float *a, float *b, float *c, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    VecAdd<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,N);

    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
}

void scudasubs(float *a, float *b, float *c, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    VecSubs<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,N);

    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

}

void scudascalmult(float w, float *a, float *b, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;

    hipMalloc(&da,size);
    hipMalloc(&db,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    ScMult<<<blocksPerGrid,threadsPerBlock>>>(w,da,db,N);

    hipMemcpy(b,db,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);

}

void scudavecmult(float *a, float *b, float *c, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    VecMult<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,N);

    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

}

void scudahamard(float *a, float *b, float *c, float *d, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    Hamard<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);

}

void scudasquarediff(float wa, float *a, float *b, float *c, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    SquareDiff<<<blocksPerGrid,threadsPerBlock>>>(wa,da,db,dc,N);

    hipMemcpy(c,dc,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);

}

void scudaorbitals(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalS<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalpx(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalPx<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalpy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalPy<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalpz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalPz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldxx(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDxx<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldxy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDxy<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldxz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDxz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldyy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDyy<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldyz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDyz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldzz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDzz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfxxx(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFxxx<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfxxy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFxxy<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfxxz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFxxz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfxyy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFxyy<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfxyz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFxyz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfxzz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFxzz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfyyy(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFyyy<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfyyz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFyyz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfyzz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFyzz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfzzz(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFzzz<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}


void scudaorbitaldy0(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDY0<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldy1(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDY1<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldy2(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDY2<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldy3(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDY3<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitaldy4(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalDY4<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfy0(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFY0<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfy1(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFY1<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfy2(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFY2<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfy3(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFY3<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfy4(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFY4<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfy5(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFY5<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}

void scudaorbitalfy6(float *a, float *b, float *c, float *d, float xs, float cx, float cy,float cz, int N)
{
    size_t size = N*sizeof(float);

    float *da;
    float *db;
    float *dc;
    float *dd;

    hipMalloc(&da,size);
    hipMalloc(&db,size);
    hipMalloc(&dc,size);
    hipMalloc(&dd,size);

    hipMemcpy(da,a,size,hipMemcpyHostToDevice);
    hipMemcpy(db,b,size,hipMemcpyHostToDevice);
    hipMemcpy(dc,c,size,hipMemcpyHostToDevice);

    int threadsPerBlock;
    int blocksPerGrid;

    if (N>M_N)
    {
        threadsPerBlock = M_N;
        blocksPerGrid = ceil( (N+threadsPerBlock-1)/threadsPerBlock);
    }

    OrbitalFY6<<<blocksPerGrid,threadsPerBlock>>>(da,db,dc,dd,xs,cx,cy,cz,N);

    hipMemcpy(d,dd,size,hipMemcpyDeviceToHost);

    hipFree(da);
    hipFree(db);
    hipFree(dc);
    hipFree(dd);
}
